#include "hip/hip_runtime.h"
//     Implementacja wielu r�wnolegle dzia�aj�cych uk�ad�w logistycznych sprz�onych s�siadami
//						na multiprocesory NVIDIA w j�zyku CUDA C
//      Wojciech Borkowski - Instytut Studi�w Spo�ecznych Uniwersytet Warszawski
//					       www.iss.uw.edu.pl/borkowski
///////////////////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <assert.h>
#include "cutil_inline.h"
#include "symshell.h" //Naglowek "prymityw�w graficznych" biblioteki SYMSHELL

struct Image {
  unsigned int 	 width;
  unsigned int 	 height;
  unsigned int 	 bytes_per_pixel; /* 3:RGB, 4:RGBA */ 
  unsigned char	 pixel_data[496 * 496 * 3];
}; 
extern "C" const Image gimp_image;

const int  LAMBDA_SELECT=1;      //Czy Lambdy losowe(0) czy inne(1) czy z obrazka(2)
const bool USE_TRUE_COLOR=true; //Czy w trybie RGB czy paletowo?
const bool SAVE_TO_FILES=false;	 //Czy zapisywać ekrany do plików?

typedef float REAL;//Nie dzia�a dla innego typu na starszych kartach (czyli wi�kszo�ci z TESL� w��cznie)
#define ALFA (REAL(3.0/10.0))//Ile bierze swojego x z poprzedniegop kroku. Od s�siad�w 1-ALFA
					//Powinno by� const a nie #define, ale wtedy nie widac w kodzie CUDA (blad kompilatora?)

size_t SIDELENGTH=31*16;//Dlugosc boku kwadratowej tablicy kom�rek - lepiej jak wielokrotno�� 16
size_t WORLDSIZE=SIDELENGTH*SIDELENGTH;//Ile jest kom�rek w sumie w kwadratowej tablicy - bo alkoacja tu tylko liniowa
size_t MAXINDEX=SIDELENGTH-1;	//Maksymalny dozwolony index

dim3 ThreadsPerBlock(4, 4); //Ile w�tk�w w ka�dym z blok�w 
dim3 Grid(SIDELENGTH / ThreadsPerBlock.x, SIDELENGTH / ThreadsPerBlock.y);//Na ile blok�w trzeba podzieli� macie�
//int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;//Jak niepodzielne przez 16
  
int MAXSTEPS=100000;//Ile zrobi� krok�w
int world_count=0;//Ile ju� zrobiono krok�w
clock_t timer=0;//Czas wykonania kernela z punktu widzenia systemu
int vis_freq=2;//max(1,min(MAXSTEPS-1,1));//Co ile krok�w wizualizacja - nie mniej ni� 1
int VSIZ=2;//Szeroko�� kwadracika - na razie nie u�ywane

//extern "C" ???
int WB_error_enter_before_clean=1;

// "Swiat" jest co prawda kwadratowy, ale jest alokowany jako liniowy blok, bo tak tylko umie CUDA
// wiec zeby myslec o nim jako o miaciezy kwadratowej trzeba recznie przeliczac funkcja getval(row,col)
//////////////////////////////////////////////////////////////////////////////////////////////////////////
REAL* worldA=NULL;//Stan aktualny/nastepny
REAL* worldB=NULL;//Stan aktualny/nastepny drugi (u�ywane na zmiane)
REAL* Lambdy=NULL;//Wartosci parametru kontroli
unsigned* Far=NULL;//Dalekie linki

__host__ __device__ inline
int Ind(int row,int col,size_t SIDELENGTH)
{
	return row*SIDELENGTH+col;
}

__host__ inline //Wersja dla glownego procesora
REAL& getval(REAL* world,int row,int col)//Zwraca referencje, wi�c sluzy do czytania i pisania
{/*
							assert(row>=0);
							assert(row<SIDELENGTH);
							assert(col>=0);
							assert(col<SIDELENGTH); */
	int index=row*SIDELENGTH+col;
	return world[index];
}

__device__ //Wersja dla CUDA
REAL& _getval(REAL* world,int row,int col,size_t SIDELENGTH)//Zwraca referencje, wi�c sluzy do czytani i pisania
{						
	int index=row*SIDELENGTH+col;
	return world[index];
}

//Funkcje dla glownego procesora do zainicjowania tablic "swiata"
///////////////////////////////////////////////////////////////////////
void setvalues(REAL* world,REAL val) //Ta sama wartosc
{
						assert(world!=NULL);
	for(size_t i=0;i<WORLDSIZE;i++)
		world[i]=val;
}

void randvalues(REAL* world) //Wartsci losowe
{
						assert(world!=NULL);
	for(size_t i=0;i<WORLDSIZE;i++)
		world[i]=double(rand())/double(RAND_MAX);
}

void initvaluesvert(REAL* world,double maxval=1) //Gradient wertykalny
{
						assert(world!=NULL);
	for(size_t j = 0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
	{
		double pom=double(j)/double(SIDELENGTH)*maxval;
		for(size_t i = 0;i<SIDELENGTH;i++)//Kolejne wiersze
		{
			getval(world,i,j)=pom;
		}
	}
}

void initvalueshoriz(REAL* world,double maxval=1) //Gradient horyzontalny
{
							assert(world!=NULL);
	for(size_t i = 0;i<SIDELENGTH;i++)//Kolejne wiersze
	{
		double pom=double(i)/double(SIDELENGTH)*maxval;
		for(size_t j = 0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
		{
			getval(world,i,j)=pom;
		}
	}
}

void initvaluesfromImage(REAL* table,const Image& the_image)
{
										assert(the_image.width==SIDELENGTH);
										assert(the_image.height==SIDELENGTH);
	for(size_t i = 0;i<SIDELENGTH;i++)//Kolejne wiersze
	{
		for(size_t j = 0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
		{
			unsigned first=(i*SIDELENGTH+j)*the_image.bytes_per_pixel;
			double pom=the_image.pixel_data[first]+the_image.pixel_data[first+1]+the_image.pixel_data[first+2];
			pom/=3;//srednia jasnosc
			pom/=255; //Zakres 0..1
			getval(table,i,j)=pom;
		}
	}
}

void initfarconnections(unsigned* Far,REAL* Lambdy)
{
								assert(Lambdy!=NULL);
								assert(Far!=NULL);
	unsigned a,b;
	for(size_t i = 0;i<SIDELENGTH;i++)//Kolejne wiersze
	{
		for(size_t j = 0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
		if(getval(Lambdy,i,j)!=0)
		{
			do {
				double angle=double(rand())/double(RAND_MAX)*M_PI*2;
				double r=(double(rand())/double(RAND_MAX))*
						 (double(rand())/double(RAND_MAX))*
						 (double(rand())/double(RAND_MAX))*
						 (double(rand())/double(RAND_MAX))*
						 (double(rand())/double(RAND_MAX))*
						 (double(rand())/double(RAND_MAX))*
						 SIDELENGTH/2;
				a=(SIDELENGTH+i+r*sin(angle));
					a%=SIDELENGTH;
				b=(SIDELENGTH+j+r*cos(angle));
					b%=SIDELENGTH;
			}while(getval(Lambdy,a,b)==0);

			line(j,i,b,a,1+(i*j)%254);
			Far[Ind(i,j,SIDELENGTH)]=a*SIDELENGTH+b;
		}
		else Far[Ind(i,j,SIDELENGTH)]=0;

		flush_plot();
	}
}

//DO SYMULACJI NA CUDA
////////////////////////////
__global__ //Definicja tego co robi jeden watek - obliczaj�cy nowy stan jednej kom�rki
void kernel_mem(REAL* old_world, REAL* world, REAL* Lambdy, unsigned* Far, size_t SIDELENGTH, unsigned int stepcount)
{   
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;//Mapowanie 'i' ze stand. struktur opisu gridu watk�w
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;//Mapowanie 'j' ze stand. struktur
	unsigned int index=Ind(i,j,SIDELENGTH);

	REAL x=old_world[index];//_getval(old_world,i,j,SIDELENGTH);
	REAL r=Lambdy[index];//_getval(Lambdy,i,j,SIDELENGTH);
	if(r<0.1) r=0;
		else	r=3.5+r/2;//sqrt(r*4);
	REAL xs=0;//Na "magazynowanie" wp�ywu s�siad�w von Neumana r=2 (krzyzyk)
			  //Nieistniejacy sasiedzi maja zawsze stan = 0
	//Bliscy sasiedzi
	unsigned int LIM=SIDELENGTH-1;
	if(i<LIM) xs+=_getval(old_world,i+1,j,SIDELENGTH);
	if(j<LIM) xs+=_getval(old_world,i,j+1,SIDELENGTH);
	if(i>0) xs+=_getval(old_world,i-1,j,SIDELENGTH);
	if(j>0) xs+=_getval(old_world,i,j-1,SIDELENGTH);
	//Troch� dalsi s�siedzi
	LIM=SIDELENGTH-2;
	if(i>1) xs+=_getval(old_world,i-2,j,SIDELENGTH);
	if(j>1) xs+=_getval(old_world,i,j-2,SIDELENGTH);
	if(i<LIM) xs+=_getval(old_world,i+2,j,SIDELENGTH);
	if(j<LIM) xs+=_getval(old_world,i,j+2,SIDELENGTH);
    //Najdalszy "s�siad" - dalekie po��czenie
	LIM=Far[index];
	
	if(LIM>0)
	{
		xs+=old_world[LIM]*2;
		xs/=10;//Sredni wp�yw od s�siad�w
	} else xs/=8;

	x=ALFA*x+(1.0-ALFA)*xs;//Realizacja wp�ywu s�siad�w
	world[index]=(1-x)*x*r;
}

//DO SYMULACJI NA G��WNYM PROCESORZE - W CELU POR�WNANIA
//////////////////////////////////////////////////////////////
__host__
void pseudo_kernel(REAL* old_world, REAL* world, REAL* Lambdy, size_t SIDELENGTH, unsigned int stepcount)
{
	for(unsigned int j = 0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
		for(unsigned int i = 0;i<SIDELENGTH;i++)//Kolejne wiersze
			//Wszystko jedno co pierwsze - gdy ca�a tablica w cashu, ale jak nie?
		{
			REAL x=getval(old_world,i,j);
			REAL r=getval(Lambdy,i,j);
			if(r<0.1) r=0;
			else	r=3+r;//sqrt(r*4);
			REAL xs=0;//Na "magazynowanie" wp�ywu s�siad�w von Neumana r=2 (krzyzyk)
					  //Nieistniejacy sasiedzi maja zawsze stan = 0
			unsigned int LIM=SIDELENGTH-1;
			if(i<LIM) xs+=getval(old_world,i+1,j);
			if(j<LIM) xs+=getval(old_world,i,j+1);
			if(i>0) xs+=getval(old_world,i-1,j);
			if(j>0) xs+=getval(old_world,i,j-1);
			LIM=SIDELENGTH-2;
			if(i>1) xs+=getval(old_world,i-2,j);
			if(j>1) xs+=getval(old_world,i,j-2);
			if(i<LIM) xs+=getval(old_world,i+2,j);
			if(j<LIM) xs+=getval(old_world,i,j+2);
			
			xs/=8;//Sredni wp�yw od s�siad�w
			x=ALFA*x+(1.0-ALFA)*xs;//Realizacja wp�ywu s�siad�w
			getval(world,i,j)=(1-x)*x*r;
		}
}

// FUNKCJE OPAKOWANIA - DO WIZUALIZACJI I STEROWANIA
///////////////////////////////////////////////////////////////////

//Drukowanie malych tablic tekstowo
void print_world(REAL* world)
{
						assert(world!=NULL);
	printf("\n\f%6d\n",world_count);
	for(unsigned  i=0;i<SIDELENGTH;i++)//KOlejne wiersze
	{
		for(unsigned j=0;j<SIDELENGTH;j++)//Kolumny-komorki w wierszu
		{
			//printf("%2d",getval(world,i,j));
			int c=32+getval(world,i,j)*64;
			printf("%c",c);
		}
		printf("!\n");
	}
}

// Wyswietlanie duzych tablic z uzyciem grafiki symshell'a
void display(REAL* world,double maxval=1,bool Yellow=0)
{
											assert(world!=NULL);
	int end=screen_width();
	if(end>SIDELENGTH)
			end=SIDELENGTH;
	printc(0,end,128,255,"%6d",world_count);
	if(Yellow)
	{
		for(int i=0;i<end;i++)//KOlejne wiersze
		{
			for(int j=0;j<end;j++)//Kolumny-komorki w wierszu
			{
				double pom=getval(world,i,j);
				pom/=maxval;
				pom*=255;
				if(!USE_TRUE_COLOR)
					plot(j,i,pom); //Wersja z domyslna paleta kolorow
				else
					plot_rgb(j,i,pom,pom,0);//Wersja w "zlotych szarosciach" RGB
			}
		}
	}
	else
	{
		for(int i=0;i<end;i++)//KOlejne wiersze
		{
			for(int j=0;j<end;j++)//Kolumny-komorki w wierszu
			{
				double pom=getval(world,i,j);
				pom/=maxval;
				pom*=255;
				if(!USE_TRUE_COLOR)
					plot(j,i,pom);//Wersja z domyslna paleta kolorow
				else
					plot_rgb(j,i,pom,0,pom);//Wersja we "fioletowych szarosciach" RGB
			}
		}
	}
	printc(0,end-char_height('1'),240,0,"%6d",world_count);
	const char* podpis="wborkowski@swps.edu.pl";
	printc(SIDELENGTH-string_width(podpis),end-string_height(podpis),256+28,0,"%s",podpis);
	flush_plot();//Obrazek juz gotowy - wyswietlic!
}


//PRINT or/and DISPLAY HELP
void Help()
{
	//...
}


//Sprawdzanie wejscia z okna graficznego - dosyc podstawowe
int check_gr_input()
{
	char tab[2];
	tab[1]=0;
	if(input_ready())//Jesli przyszedl jakis komunikat
	{
		tab[0]=get_char();
		switch(tab[0])
		{
			case '@'://Wymuszony replot ekranu - tu drukowanie Lambd
			case '\r':display(Lambdy,1,true);break;
			case 'q'://Przerwanie aktualnej funkcji symulacji lub wyjscie z programu
			case EOF:return -1;
			case 'h'://Drukowanie helpu jak przyszedl niezdefiniowany w switch'u komunikat
			default:
					Help();break;
		}
	}
	//DOMYSLNE
	return 0; //Jak nie przyszedl lub zostal poprawnie obsluzony
}


//"PROCEDURY" STEROWANIA WYKONANIEM SYMULACJI NA CUDA'IE i NA GLOWNYM CPU
////////////////////////////////////////////////////////////////////////////

void run_on_device() //Wykonanie na CUDA
{
								assert(worldA!=NULL);  //Swiat w glownej pamieci musi byc juz gotowy 
								assert(worldB!=NULL);	
								assert(Lambdy!=NULL);
								assert(Far!=NULL);
	//Wskazniki do wersji/kopii swiata w pamieci karty [obliczeniowej]
	REAL* worldAdev=NULL;
	REAL* worldBdev=NULL;
	REAL* LambdyDev=NULL;
	unsigned* FarDev=NULL;

	// Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&worldAdev, WORLDSIZE*sizeof(worldAdev[0]) ) );
    cutilSafeCall( hipMalloc((void**)&worldBdev, WORLDSIZE*sizeof(worldBdev[0]) ) );
	cutilSafeCall( hipMalloc((void**)&LambdyDev, WORLDSIZE*sizeof(LambdyDev[0]) ) );
	cutilSafeCall( hipMalloc((void**)&FarDev, WORLDSIZE*sizeof(FarDev[0]) ) );
	// Copy vector A from host memory to device memory
    cutilSafeCall( hipMemcpy(worldAdev, worldA, WORLDSIZE*sizeof(worldAdev[0]), hipMemcpyHostToDevice) );	
	// Copy table of parameters (Lambdy) into device memory
	cutilSafeCall( hipMemcpy(LambdyDev, Lambdy, WORLDSIZE*sizeof(LambdyDev[0]), hipMemcpyHostToDevice) );
	// Copy table of far connections
	cutilSafeCall( hipMemcpy(FarDev, Far, WORLDSIZE*sizeof(FarDev[0]), hipMemcpyHostToDevice) );

	//DEBUG ONLY - z powrotem do g�ownej pamieci dla sprawdzenia
	//cutilSafeCall( hipMemcpy(worldA,worldAdev, WORLDSIZE*sizeof(worldAdev[0]), hipMemcpyDeviceToHost) );
	
	if(SIDELENGTH<64) //Co� na EKRANIK TEKSTOWY - jesli si� zmiesci
		print_world(worldA);

	display(worldA); //Stan poczatkowy na ekranie graficznym

	//Poczatkowe przypisanie "swiatow" w pamieci karty
	REAL* old_world=worldAdev;
	REAL* new_world=worldBdev;
	
	timer=clock(); //Poczatek liczenia sumarycznego czasu wykonania wzgledem procesora glownego - dziala tylko zgrubnie 

	//PETLA SYMULACJI NA CUDA:
    while(world_count < MAXSTEPS) 
    {
		// Run one step calculation on device - Grid & ThreadsPerBlock defined much obove
        kernel_mem<<< Grid,ThreadsPerBlock >>>(old_world, new_world, LambdyDev, FarDev, SIDELENGTH,world_count+1);
        cutilCheckMsg("kernel_mem launch failure");

		//Kernel jest wzywany asynchronicznie, wi�c "za darmo" mo�na co� tymczasem sprawdzi� i nawet mo�e odrysowa�
		if(check_gr_input()==-1) goto KONIEC_PIESNI;//Moze "skoczyc do wyjscia"
		
#ifdef DEBUG
		cutilSafeCall( hipDeviceSynchronize() );//Zeby dzialalo poprawnie debugowanie to trzeba poczekac az skonczy
#endif

		++world_count;//Zliczanie krok�w symulacji
		
		if(world_count%vis_freq==0) //Sci�gniecie danych czasochlonne - wiec tylko dla niektorych krokow jest wizualizacja
		{	
			// Copy result from device memory to host memory for visualisation (SYNCHRONISATION BY DEFAULT)
			cutilSafeCall( hipMemcpy(worldB,new_world, WORLDSIZE*sizeof(worldB[0]), hipMemcpyDeviceToHost) );
			if((SIDELENGTH<64)) print_world(worldB);//Cos na konsole tekstowa
				//else printf("\r%4d",world_count);
			display(worldB);//Wyswietlenie danych z poprzedzajacego czytania
			if(SAVE_TO_FILES)
			{
				char Filename[256];
				sprintf(Filename,"LoSyn%08d",world_count);
				dump_screen(Filename);
			}
		}
		
		//Preparing for next step - swaping vectors
		REAL* pom=old_world;
		old_world=new_world;
		new_world=pom;
    }

KONIEC_PIESNI:
	printf("\n%g ms/step",(clock()-timer)/double(world_count));//Podliczenie sumarycznego czasu wykonania

    // Niezbedne porzadki na karcie (free device memory)
    if (worldAdev)
        hipFree(worldAdev);
    if (worldBdev)
        hipFree(worldBdev);
	if(LambdyDev)
		hipFree(LambdyDev);
	if(FarDev)
		hipFree(FarDev);
}

void run_on_host( ) //Dla porownania wykonanie na procesorze glownym
{
								assert(worldA!=NULL);
								assert(worldB!=NULL);		
	if((SIDELENGTH<64)) print_world(worldA);//Co� na EKRANIK TEKSTOWY
	display(worldA); //Stan poczatkowy na ekranie graficznym

	REAL* old_world=worldA;//Poczatkowe przypisanie "swiatow"
	REAL* new_world=worldB;//... w pamieci gl�wnej (CPU)
	
	timer=clock();//Poczatek liczenia sumarycznego czasu wykonania wzgledem procesora glownego

	//PETLA SYMULACJI NA CPU:
    while(world_count < MAXSTEPS) 
    {
		// Run calculation on main CPU
        pseudo_kernel(old_world, new_world, Lambdy, SIDELENGTH,world_count+1);

		if(check_gr_input()==-1) goto KONIEC_PIESNI;//Moze "skoczyc do wyjscia"

        ++world_count;//Zliczanie krok�w symulacji
		
		if(world_count%vis_freq==0)//Wizualizacja tez tylko od czasu do czasu
		{
			if((SIDELENGTH<64)) print_world(new_world);
			else printf("\r%4d",world_count);
			display(new_world);
		}
		
		//Preparing for next step - swaping vectors and increment counters
		REAL* pom=old_world;
		old_world=new_world;
		new_world=pom;
    }
KONIEC_PIESNI:
	printf("\n%g ms/step",(clock()-timer)/double(world_count));//Podliczenie czasu i juz
}

// Main host code
///////////////////////////////////////
int main(int argc, const char** argv)
{
	//Gdyby chcie� zaczyna� z warto�ci losowych
	srand(time(NULL));//No chyba, �e by si� chcia�o powtarzalnie debugowa� wtedy jaka� liczba zamias time(NULL)
	//srand(10);

	mouse_activity(0);   /* Mysz w obrebie okna nie bedzie potrzebna */
	set_background(255); /* Biale tlo */
	buffering_setup(1);  /* Wlaczone buforowanie do animacji - widac dopiero jak obrazek gotowy */
	shell_setup("Logistic iterations - synchronisation",argc,argv); /* Tytul okna i przekazanie parametr�w wywolania */

    if(!init_plot(SIDELENGTH,SIDELENGTH,0,0)) //Otwarcie okna z miejscem na g��wn� wizualizacj� swiata i jakie� boki
	{
		fprintf(stderr,"Can't initialize graphics\n");//Jak sie nie udalo...		
		exit(-11);
	}

	printf("\nSpace of coupled logistic iterators %dx%d (%d = %d bajts)\n",SIDELENGTH,SIDELENGTH,WORLDSIZE,WORLDSIZE*sizeof(REAL));
	//Alokacja tablic "Swiat" w glownej pamieci
	worldA=new REAL[WORLDSIZE];
	worldB=new REAL[WORLDSIZE];
	Lambdy=new REAL[WORLDSIZE];
	Far=new unsigned[WORLDSIZE];
	
	//Test symulacji na CUDA
	//////////////////////////////
	printf("\nCalculation on CUDA compatible card (REAL):\n");
	world_count=0;//Ile ju� zrobiono krok�w
	//initvalueshoriz(worldA);//ALTERNATYWNIE
	randvalues(worldA);//ALTERNATYWNIE
	switch(LAMBDA_SELECT){
	default:
	case 0: randvalues(Lambdy);break;//ALTERNATYWNIE
	case 1: initvaluesvert(Lambdy);break;//ALTERNATYWNIE
	case 2: initvaluesfromImage(Lambdy,gimp_image);break;
    }
	initfarconnections(Far,Lambdy);
	setvalues(worldB,0);//Zerowanie tablicy na pierwsze wyniki
	run_on_device(); //WLASCIWA SYMULACJA
/*
	//Test symulacji na jednym rdzeniu CPU
	////////////////////////////////////////////////////
	printf("\nCalculation on one core of main CPU:\n");
	world_count=0;//Ile ju� zrobiono krok�w
	//initvalueshoriz(worldA);
	initvaluesvert(Lambdy);
	setvalues(worldB,0);
	run_on_host();
*/
	//Porzadki na zakonczenie calego programu
	delete worldA;//zwalnianie pamieci
	delete worldB;//zwalnianie pamieci
	delete Lambdy;//zwalnianie pamieci
	delete Far;//zwalnianie pamieci
	
	close_plot(); //Zamkniecie okna graficznego

	return 0;
}

/* 
__global__ //Kernel testowy do sprawdzania czy bloki i watki sie poprawnie mapuja
void kernel_test(REAL* old_world, REAL* world, REAL* Lambdy, size_t SIDELENGTH, unsigned int stepcount)
{
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	
	int index=j*SIDELENGTH+i;
	//world[index]=index;
	
	if(stepcount==1)
		world[index]=threadIdx.x;//TEST 1
	else
	if(stepcount==2)
		world[index]=threadIdx.y;//TEST 1B
	else
	if(stepcount==3)
		world[index]=blockIdx.x;//TEST 2
	else
	if(stepcount==4)
		world[index]=blockIdx.y;//TEST 2B
	else
	if(stepcount==5)
		world[index]=i;//TEST 3
	else
	if(stepcount==6)
		world[index]=j;//TEST 3B
	else
		world[index]=blockDim.x;//TEST 4
	//world[index]=blockIdx.y;//TEST 4B
	//
}
*/
